#include "hip/hip_runtime.h"
#include <cmath>
#include <memory>
#include <stdexcept>
#include <string>
#include <utility>

#include <type_traits>

#include "ArgumentParser/ArgumentParser.hpp"
#include "FilenamesParser/FilenamesParser.h"
#include "PoppyBuilder/PoppyBuilder.h"
#include "PositionsBuilder/ContinuousPositionsBuilder.hpp"
#include "Presearcher/Presearcher.cuh"
#include "ResultsPrinter/ContinuousResultsPrinter.hpp"
#include "SbwtBuilder/SbwtBuilder.h"
#include "SbwtContainer/GpuSbwtContainer.cuh"
#include "SbwtContainer/SbwtContainer.h"
#include "Searcher/ContinuousSearcher.cuh"
#include "SeqToBitsConverter/ContinuousSeqToBitsConverter.hpp"
#include "SequenceFileParser/ContinuousSequenceFileParser.h"
#include "Utils/BenchmarkUtils.hpp"
#include "Utils/CudaUtils.cuh"
#include "Utils/Logger.h"
#include "Utils/MemoryUtils.hpp"
#include "Utils/TypeDefinitions.h"
#include "fmt/core.h"

using memory_utils::get_total_system_memory;
using std::remove_reference;
using std::runtime_error;
using std::string;
using namespace sbwt_search;
using fmt::format;
using gpu_utils::get_free_gpu_memory;
using log_utils::Logger;
using math_utils::round_down;
using std::endl;
constexpr auto WARN = Logger::LOG_LEVEL::WARN;
constexpr auto INFO = Logger::LOG_LEVEL::INFO;
constexpr auto DEBUG = Logger::LOG_LEVEL::DEBUG;

auto get_gpu_container(string index_file) -> shared_ptr<GpuSbwtContainer>;
auto get_max_chars_per_batch(
  size_t unavailable_memory, uint max_batches, size_t max_cpu_memory
) -> size_t;
auto get_max_chars_per_batch_gpu(uint max_batches) -> size_t;
auto get_max_chars_per_batch_cpu(
  size_t unavailable_memory, uint max_batches, size_t max_memory
) -> size_t;

const auto program_name = "SBWT Search";
const auto program_description
  = "An application to search for k-mers in a genome given an SBWT index";
const uint kmer_size = 30;
const auto num_seq_to_bit_converters = 3;

auto main(int argc, char **argv) -> int {
  Logger::initialise_global_logging(WARN);
  Logger::log_timed_event("main", Logger::EVENT_STATE::START);
  Logger::log_timed_event("SBWTLoader", Logger::EVENT_STATE::START);
  auto args = ArgumentParser(program_name, program_description, argc, argv);
  auto gpu_container = get_gpu_container(args.get_index_file());
  Logger::log_timed_event("SBWTLoader", Logger::EVENT_STATE::STOP);
  FilenamesParser filenames_parser(
    args.get_sequence_file(), args.get_output_file()
  );
  auto input_filenames = filenames_parser.get_input_filenames();
  auto output_filenames = filenames_parser.get_output_filenames();
  const auto max_batches = args.get_batches();
  const auto max_chars_per_batch = get_max_chars_per_batch(
    args.get_unavailable_ram(), max_batches, args.get_max_cpu_memory()
  );
  if (max_chars_per_batch == 0) { throw runtime_error("Not enough memory"); }
  Logger::log(
    INFO, "Using " + to_string(max_chars_per_batch) + " characters per batch"
  );
  omp_set_nested(1);
#pragma omp parallel
#pragma omp single
  Logger::log(
    INFO, format("Running OpenMP with {} threads", omp_get_num_threads())
  );
  using SequenceFileParser = ContinuousSequenceFileParser;
  auto sequence_file_parser = make_shared<SequenceFileParser>(
    input_filenames,
    kmer_size,
    max_chars_per_batch,
    max_chars_per_batch,
    num_seq_to_bit_converters,
    max_batches
  );
  using SeqToBitsConverter
    = ContinuousSeqToBitsConverter<ContinuousSequenceFileParser>;
  auto seq_to_bit_converter = make_shared<SeqToBitsConverter>(
    sequence_file_parser,
    num_seq_to_bit_converters,
    kmer_size,
    max_chars_per_batch,
    max_batches
  );
  using PositionsBuilder
    = ContinuousPositionsBuilder<ContinuousSequenceFileParser>;
  auto positions_builder = make_shared<PositionsBuilder>(
    sequence_file_parser, kmer_size, max_chars_per_batch, max_batches
  );
  using Searcher = ContinuousSearcher<PositionsBuilder, SeqToBitsConverter>;
  auto searcher = make_shared<Searcher>(
    gpu_container,
    seq_to_bit_converter,
    positions_builder,
    max_batches,
    max_chars_per_batch
  );
  using ResultsPrinter = ContinuousResultsPrinter<
    Searcher,
    SequenceFileParser,
    SeqToBitsConverter>;
  auto results_printer = make_shared<ResultsPrinter>(
    searcher,
    sequence_file_parser,
    seq_to_bit_converter,
    output_filenames,
    kmer_size
  );
#pragma omp parallel sections default(shared)
  {
#pragma omp section
    { sequence_file_parser->read_and_generate(); }
#pragma omp section
    { seq_to_bit_converter->read_and_generate(); }
#pragma omp section
    { positions_builder->read_and_generate(); }
#pragma omp section
    { searcher->read_and_generate(); }
#pragma omp section
    { results_printer->read_and_generate(); }
  }
  Logger::log(INFO, "DONE");
  Logger::log_timed_event("main", Logger::EVENT_STATE::STOP);
}

auto get_gpu_container(string index_file) -> shared_ptr<GpuSbwtContainer> {
  Logger::log_timed_event("SBWTParserAndIndex", Logger::EVENT_STATE::START);
  auto builder = SbwtBuilder(index_file);
  auto cpu_container = builder.get_cpu_sbwt(true);
  Logger::log_timed_event("SBWTParserAndIndex", Logger::EVENT_STATE::STOP);
  Logger::log_timed_event("SBWT_GPU_Transfer", Logger::EVENT_STATE::START);
  auto gpu_container = cpu_container->to_gpu();
  Logger::log_timed_event("SBWT_GPU_Transfer", Logger::EVENT_STATE::STOP);
  auto presearcher = Presearcher(gpu_container);
  Logger::log_timed_event("Presearcher", Logger::EVENT_STATE::START);
  presearcher.presearch();
  Logger::log_timed_event("Presearcher", Logger::EVENT_STATE::STOP);
  return gpu_container;
}

auto get_max_chars_per_batch(
  size_t unavailable_memory, uint max_batches, size_t max_cpu_memory
) -> size_t {
  auto gpu_chars = get_max_chars_per_batch_gpu(max_batches);
  auto cpu_chars = get_max_chars_per_batch_cpu(
    unavailable_memory, max_batches, max_cpu_memory
  );
  if (gpu_chars < cpu_chars) { return gpu_chars; }
  return cpu_chars;
}

auto get_max_chars_per_batch_gpu(uint max_batches) -> size_t {
  size_t free = get_free_gpu_memory();
  auto max_chars_per_batch
    = round_down<size_t>(free * 8 / 66 / max_batches, threads_per_block);
  Logger::log(
    Logger::LOG_LEVEL::DEBUG,
    format(
      "Free gpu memory: {} bits ({:.2f}GB). This allows for {} characters per "
      "batch",
      free,
      double(free) / 1024 / 1024 / 1024,
      max_chars_per_batch
    )
  );
  return max_chars_per_batch;
}

auto get_max_chars_per_batch_cpu(
  size_t unavailable_memory, uint max_batches, size_t max_memory
) -> size_t {
  if (unavailable_memory > get_total_system_memory() * 8) {
    throw runtime_error("Not enough memory. Please specify a lower number of "
                        "unavailable-main-memory.");
  }
  size_t free = get_total_system_memory() * 8 - unavailable_memory;
  if (max_memory < free) { free = max_memory; }
  auto max_chars_per_batch
    = round_down<size_t>(free / 460 / (max_batches), threads_per_block);
  Logger::log(
    DEBUG,
    format(
      "Free main memory: {} bits ({:.2f}GB). This allows for {} "
      "characters per batch",
      free,
      double(free) / 8 / 1024 / 1024 / 1024,
      max_chars_per_batch
    )
  );
  return max_chars_per_batch;
}
