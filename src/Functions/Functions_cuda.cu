#include "hip/hip_runtime.h"
#include "CudaUtils.cuh"
#include "Functions.h"

namespace functions {

__global__ auto device_add(int *a, int *b, int *out, size_t n) -> void;
__global__ auto device_mul(int *a, int *b, int *out, size_t n) -> void;

auto add(int a, int b) -> int {
  int *device_a = nullptr, *device_b = nullptr;
  hipMalloc((void **)&device_a, sizeof(int));
  hipMalloc((void **)&device_b, sizeof(int));
  hipMemcpy(device_a, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_b, &b, sizeof(int), hipMemcpyHostToDevice);
  device_add<<<1, 1024>>>(device_a, device_b, device_a, 1);
  hipMemcpy(&a, device_a, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(device_a);
  hipFree(device_b);
  return a;
}

__global__ auto device_add(int *a, int *b, int *out, size_t n) -> void {
  auto idx = get_idx();
  if (idx < n) { out[idx] = a[idx] + b[idx]; }
}

auto mul(int a, int b) -> int {
  int *device_a = nullptr, *device_b = nullptr;
  hipMalloc((void **)&device_a, sizeof(int));
  hipMalloc((void **)&device_b, sizeof(int));
  hipMemcpy(device_a, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_b, &b, sizeof(int), hipMemcpyHostToDevice);
  device_mul<<<1, 1024>>>(device_a, device_b, device_a, 1);
  hipMemcpy(&a, device_a, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(device_a);
  hipFree(device_b);
  return a;
}

__global__ auto device_mul(int *a, int *b, int *out, size_t n) -> void {
  auto idx = get_idx();
  if (idx < n) { out[idx] = a[idx] * b[idx]; }
}

}
