#include "hip/hip_runtime.h"
#include "ColorSearcher/ColorPostProcessor.cuh"
#include "ColorSearcher/ColorSearcher.cuh"
#include "ColorSearcher/ColorSearcher.h"
#include "Tools/BitDefinitions.h"
#include "Tools/GpuUtils.h"
#include "Tools/Logger.h"
#include "Tools/MathUtils.hpp"
#include "fmt/core.h"
#include "hip/hip_runtime.h"

namespace sbwt_search {

using bit_utils::set_bits;
using fmt::format;
using log_utils::Logger;
using math_utils::divide_and_ceil;

auto ColorSearcher::launch_search_kernel(u64 num_queries, u64 batch_id)
  -> void {
  Logger::log_timed_event(
    format("SearcherSearch_{}", stream_id),
    Logger::EVENT_STATE::START,
    format("batch {}", batch_id)
  );
  u64 blocks_per_grid = divide_and_ceil<u64>(num_queries, threads_per_block);
  start_timer.record(&gpu_stream);
  hipLaunchKernelGGL(
    d_color_search,
    blocks_per_grid,
    threads_per_block,
    0,
    *static_cast<hipStream_t *>(gpu_stream.get()),
    d_sbwt_index_idxs.get(),
    container->key_kmer_marks.get(),
    container->key_kmer_marks_poppy_layer_0.get(),
    container->key_kmer_marks_poppy_layer_1_2.get(),
    container->color_set_idxs.get(),
    container->color_set_idxs_width,
    set_bits.at(container->color_set_idxs_width),
    container->is_dense_marks.get(),
    container->is_dense_marks_poppy_layer_0.get(),
    container->is_dense_marks_poppy_layer_1_2.get(),
    container->dense_arrays.get(),
    container->dense_arrays_intervals.get(),
    container->dense_arrays_intervals_width,
    set_bits.at(container->dense_arrays_intervals_width),
    container->sparse_arrays.get(),
    container->sparse_arrays_width,
    set_bits.at(container->sparse_arrays_width),
    container->sparse_arrays_intervals.get(),
    container->sparse_arrays_intervals_width,
    set_bits.at(container->sparse_arrays_intervals_width),
    container->num_colors,
    d_fat_results.get()
  );
  end_timer.record(&gpu_stream);
  GPU_CHECK(hipPeekAtLastError());
  GPU_CHECK(hipStreamSynchronize(*static_cast<hipStream_t *>(gpu_stream.get()))
  );
  float millis = start_timer.time_elapsed_ms(end_timer);
  Logger::log(
    Logger::LOG_LEVEL::DEBUG,
    format("Batch {} took {} ms to search in the GPU", batch_id, millis)
  );
  Logger::log_timed_event(
    format("SearcherSearch_{}", stream_id),
    Logger::EVENT_STATE::STOP,
    format("batch {}", batch_id)
  );
}

auto ColorSearcher::launch_combine_kernel(
  u64 num_warps, u64 num_reads, u64 num_colors, u64 batch_id
) -> void {
  Logger::log_timed_event(
    format("SearcherPostProcess_{}", stream_id),
    Logger::EVENT_STATE::START,
    format("batch {}", batch_id)
  );
  u64 blocks_per_grid
    = divide_and_ceil<u64>(num_reads * num_colors, threads_per_block);
  auto &d_warps_before_new_read = d_sbwt_index_idxs;
  hipLaunchKernelGGL(
    d_post_process,
    blocks_per_grid,
    threads_per_block,
    0,
    *static_cast<hipStream_t *>(gpu_stream.get()),
    d_fat_results.get(),
    d_warps_before_new_read.get(),
    num_warps,
    num_reads,
    num_colors,
    d_results.get()
  );
  Logger::log_timed_event(
    format("SearcherPostProcess_{}", stream_id),
    Logger::EVENT_STATE::STOP,
    format("batch {}", batch_id)
  );
}

}  // namespace sbwt_search
